#include "hip/hip_runtime.h"
#include<cstdio>

extern "C" {
__global__
void transpose(int* src, int* dst){

	__shared__ int cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 

}

void oneDimSobel(unsigned char *src, unsigned char *dst, int*N){
	int n = N[0]; // lines
	int m = N[1]; // columns
	int SIZE = n*m;
	int th_x = blockIdx.x*1024+threadIdx.x;
	if(th_x < m){
		int prev = src[th_x];
		int next=-1;
		for(int i=th_x+m; i<SIZE; i+=m){
			next = src[i];
			int temp = next-prev;
			temp = temp>=0?temp:-temp;
			dst[i] = temp;
		}
	}
}
/*__global__
void columning(int* cc, int* bitmap, int* N, int* myBool){
	int n =  *N;
	int SIZE =  n * n;
    int th_x = blockIdx.x * 1024 + threadIdx.x;
	//printf("thx = %d\n", th_x);
	if(th_x < n){
		int i = th_x+n;
		//printf("nums %d %d\n", i, i-n);
		for(i = th_x+n; i<SIZE; i+=n){
			if(bitmap[i]==bitmap[i-n] && cc[i-n]<cc[i]){
				cc[i] = cc[i-n];
				*myBool = 1;
			}
		}
		i-=n;
		for(i = i-n ;i>=0; i-=n){
			if(bitmap[i]==bitmap[i+n] && cc[i+n]<cc[i]){
				cc[i]=cc[i+n];
				*myBool = 1;
			}
		}
	}
}*/
}


