
#include <hip/hip_runtime.h>
#include<cstdio>
#define S 64
#define ZERO 0

extern "C" {



__device__
bool btwn(int a, int x, int y){
	return (a>=x && a<y);
}

__device__
void load_to_shared(int* src, int cache[][S], int th_x, int th_y, int n, int m){
	int val, pos, ind_x, ind_y;
	if(threadIdx.x==0 && threadIdx.y==0){
		ind_x = th_x-1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x] = val;
	}
	
	if(threadIdx.x==0 && threadIdx.y==31){
		ind_x = th_x-1; ind_y = th_y+1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==0){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+2] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==31){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+2] = val;
	}

	if(threadIdx.y==0){
		ind_x = th_x; ind_y = th_y-1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+1] = val;
	}

	if(threadIdx.y==31){
		ind_x = th_x; ind_y = th_y+1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+1] = val;
	}
	
	if(threadIdx.x==0){
		ind_x = th_x-1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x] = val;
	}

	if(threadIdx.x==31){
		ind_x = th_x+1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x+2] = val;
	}
}

__global__
void sobelAndSuppression(int* src, int* dst_magn){
	__shared__ int cache[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_x, ind_y;

	/*now we load to share with a frame of thickness eq 1*/
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; //it's correct position
	int mag_x;
	int mag_y;
	__syncthreads();
	mag_x = cache[ind_y][ind_x-1] - cache[ind_y][ind_x+1];
	int magAbs_x = ((mag_x>0)?mag_x:-mag_x);
	mag_y = cache[ind_y+1][ind_x] - cache[ind_y-1][ind_x];
	int magAbs_y = ((mag_y>0)?mag_y:-mag_y);
	dst_magn[i_src] = magAbs_x+magAbs_y;
	if(threadIdx.x==0 && threadIdx.y==0){
		printf("%d\n", dst_magn[i_src]);
	
	}
}

}


