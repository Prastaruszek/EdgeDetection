
#include <hip/hip_runtime.h>
#include<cstdio>
#define S 64
#define ZERO 0
#define PI 3.14159265
#define LOW 16
#define HIGH 32
#define QUEUE_SIZE 128
extern "C" {
__device__
bool btwn(int a, int x, int y){
	return (a>=x && a<y);
}

__device__
void load_to_shared(int* src, int cache[][S], int th_x, int th_y, int n, int m){
	int val, pos, ind_x, ind_y;
	if(threadIdx.x==0 && threadIdx.y==0){
		ind_x = th_x-1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x] = val;
	}
	
	if(threadIdx.x==0 && threadIdx.y==31){
		ind_x = th_x-1; ind_y = th_y+1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==0){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+2] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==31){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+2] = val;
	}

	if(threadIdx.y==0){
		ind_x = th_x; ind_y = th_y-1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+1] = val;
	}

	if(threadIdx.y==31){
		ind_x = th_x; ind_y = th_y+1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+1] = val;
	}
	
	if(threadIdx.x==0){
		ind_x = th_x-1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x] = val;
	}

	if(threadIdx.x==31){
		ind_x = th_x+1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x+2] = val;
	}
}

__global__
void sobelAndSuppression(int* src, int* dst_magni, float * tangesOut){
	__shared__ int cache[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_x, ind_y;

	/*now we load to share with a frame of thickness eq 1*/
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; //it's correct position
	int mag_x;
	int mag_y;
	__syncthreads();
	mag_x = cache[ind_y][ind_x-1] - cache[ind_y][ind_x+1];
	int magAbs_x = ((mag_x>0)?mag_x:-mag_x);
	mag_y = cache[ind_y+1][ind_x] - cache[ind_y-1][ind_x];
	int magAbs_y = ((mag_y>0)?mag_y:-mag_y);
	dst_magni[i_src] = magAbs_x+magAbs_y;
	//dst_magni[i_src] = (magAbs_x+magAbs_y)>32?255:0;
	tangesOut[i_src] = atan2((float) mag_y,(float) mag_x)*180/PI;
	if(threadIdx.x==0 && threadIdx.y==0){
		printf("Magnitude=%d x=%f angle=%f\n", dst_magni[i_src], ((float) mag_x)/((float) mag_y), tangesOut[i_src]);
	}
}

__global__
void nonMaximalSupression(int * magn, float * tanges, int * dest) {
	__shared__ int cacheMagn[34][S], cacheTanges[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_x, ind_y;

	cacheMagn[threadIdx.y+1][threadIdx.x+1] = magn[i_src];
	load_to_shared(magn, cacheMagn, th_x, th_y, n, m);
    
    //cacheTanges[threadIdx.y+1][threadIdx.x+1] = tanges[i_src];
    //load_to_shared(tanges, cacheTanges, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; 
	__syncthreads();
    float angle = tanges[i_src];
    if (angle < 0) angle = 360 + angle;
    //north && south
    int centerCell = cacheMagn[ind_y][ind_x];
    dest[i_src] = centerCell;
    if ((337.5 <= angle || angle < 22.5) || 
            (157.25 <= angle && angle < 202.5)) {
        if (cacheMagn[ind_y][ind_x+1] > centerCell ||
                cacheMagn[ind_y][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // north-east && south-west 
    else if ((22.5 <= angle && angle < 67.5) ||
            (202.5 <= angle && angle < 247.5)) {
        if (cacheMagn[ind_y-1][ind_x+1] > centerCell || 
                cacheMagn[ind_y+1][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // west && east
    else if ((67.5 <= angle && angle < 112.5) ||
                (247.5 <= angle && angle < 292.5)) {
        if (cacheMagn[ind_y+1][ind_x] > centerCell ||
                cacheMagn[ind_y-1][ind_x] > centerCell) 
                    dest[i_src] = 0;
    } // west-north && east-south
    else if ((112.5 <= angle && angle < 157.5) ||
        (292.5 <= angle || angle < 337.5)) {
        if (cacheMagn[ind_y-1][ind_x-1] > centerCell || 
                cacheMagn[ind_y+1][ind_x+1] > centerCell) 
                    dest[i_src] = 0;
    }

}	

__global__
void oneBfs(int* src, int* dst, int* changed){
	__shared__ int cache[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);
	//we can proceed
	int ind_y = threadIdx.y+1;
	int ind_x = threadIdx.x+1; //it's correct position
	//tu moze dochodzic do malych runtime errorow, jesli rozmiar bedzie za maly
	int queue[QUEUE_SIZE];
	int beg=0, end=0;
	int val = cache[ind_y][ind_x];
	if(val!=0 && val!=-1 && val!=-2){
		if(val < LOW){
			val=(cache[ind_y][ind_x]=0);
		}
		else if(val >= HIGH){
			val=(cache[ind_y][ind_x]=-2);
		}
		else{
			val=(cache[ind_y][ind_x]=-1);
		}
		*changed=1;
	}
	__syncthreads();
	int procInd_x;
	int procInd_y;
	if(val==-1){
		for(int i=-1; i<2; ++i){
			for(int j=-1; j<2; ++j){
				procInd_x = ind_x+i;
				procInd_y = ind_y+j;
				if(cache[procInd_y][procInd_x]==-2){
					//two values are near
					queue[end++] = ind_y;
					queue[end++] = ind_x;
					cache[ind_y][ind_x]=-2;
					*changed=1;
				}
			}
		}
	}
	int x_new;
	int y_new;
	
	
	while(beg!=end){
		procInd_y = queue[beg++];
		procInd_x = queue[beg++];
		for(int i=-1; i<2; ++i){
			for(int j=-1; j<2; ++j){
				x_new = procInd_x+i;
				y_new = procInd_y+j;
				if(cache[y_new][x_new]==-1 && btwn(y_new, 1, 33) 
											&& btwn(x_new, 1, 33)){
					//two values are near
					queue[end++] = y_new;
					queue[end++] = x_new;
					cache[y_new][x_new]=-2;
				}
			}
		}
	}
	__syncthreads();
	dst[i_src] = cache[ind_y][ind_x];
	
}
__global__ 
void final_battle(int* src){
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int val = src[i_src];
	if(val==-2){
		src[i_src] = 255;
	}
	else{
		src[i_src] = 0;
	}
}

__global__
void gaussianFilter(int * src, int * dest) {
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	
}

}
