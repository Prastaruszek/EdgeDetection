
#include <hip/hip_runtime.h>
#include<cstdio>
#define S 64
#define ZERO 0
#define PI 3.14159265
extern "C" {
__device__
bool btwn(int a, int x, int y){
	return (a>=x && a<y);
}

__device__
void load_to_shared(int* src, int cache[][S], int th_x, int th_y, int n, int m){
	int val, pos, ind_x, ind_y;
	if(threadIdx.x==0 && threadIdx.y==0){
		ind_x = th_x-1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x] = val;
	}
	
	if(threadIdx.x==0 && threadIdx.y==31){
		ind_x = th_x-1; ind_y = th_y+1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==0){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+2] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==31){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+2] = val;
	}

	if(threadIdx.y==0){
		ind_x = th_x; ind_y = th_y-1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+1] = val;
	}

	if(threadIdx.y==31){
		ind_x = th_x; ind_y = th_y+1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+1] = val;
	}
	
	if(threadIdx.x==0){
		ind_x = th_x-1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x] = val;
	}

	if(threadIdx.x==31){
		ind_x = th_x+1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x+2] = val;
	}
}

__global__
void sobelAndSuppression(int* src, int* dst_magni, float * tangesOut){
	__shared__ int cache[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_x, ind_y;

	/*now we load to share with a frame of thickness eq 1*/
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; //it's correct position
	int mag_x;
	int mag_y;
	__syncthreads();
	mag_x = cache[ind_y][ind_x-1] - cache[ind_y][ind_x+1];
	int magAbs_x = ((mag_x>0)?mag_x:-mag_x);
	mag_y = cache[ind_y+1][ind_x] - cache[ind_y-1][ind_x];
	int magAbs_y = ((mag_y>0)?mag_y:-mag_y);
	dst_magni[i_src] = magAbs_x+magAbs_y;
	//dst_magni[i_src] = (magAbs_x+magAbs_y)>32?255:0;
	tangesOut[i_src] = atan2((float) mag_x,(float) mag_y)*180/PI;
	if(threadIdx.x==0 && threadIdx.y==0){
		printf("Magnitude=%d x=%f angle=%f\n", dst_magni[i_src], ((float) mag_x)/((float) mag_y), tangesOut[i_src]);
	}
}

__global__
void nonMaximalSupression(int * magn, float * tanges, int * dest) {
	__shared__ int cacheMagn[34][S], cacheTanges[34][S];
	int m = gridDim.x*32;
	int n = gridDim.y*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_x, ind_y;

	cacheMagn[threadIdx.y+1][threadIdx.x+1] = magn[i_src];
	load_to_shared(magn, cacheMagn, th_x, th_y, n, m);
    
    //cacheTanges[threadIdx.y+1][threadIdx.x+1] = tanges[i_src];
    //load_to_shared(tanges, cacheTanges, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; 
	__syncthreads();
    float angle = tanges[i_src];
    if (angle < 0) angle = 360 + angle;
    //north && south
    int centerCell = cacheMagn[ind_y][ind_x];
    dest[i_src] = centerCell;
    if ((337.5 <= angle && angle < 22.5) || 
            (157.25 <= angle && angle < 202.5)) {
        if (cacheMagn[ind_y+1][ind_x] > centerCell || 
                cacheMagn[ind_y-1][ind_x] > centerCell) 
                    dest[i_src] = 0;
    } // north-east && south-west 
    else if ((22.5 <= angle && angle < 67.5) ||
            (202.5 <= angle && angle < 247.5)) {
        if (cacheMagn[ind_y+1][ind_x+1] > centerCell || 
                cacheMagn[ind_y-1][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // west && east
    else if ((67.5 <= angle && angle < 112.5) ||
                (247.5 <= angle && angle < 292.5)) {
        if (cacheMagn[ind_y][ind_x+1] > centerCell || 
                cacheMagn[ind_y][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // west-north && east-south
    else if ((112.5 <= angle && angle < 157.5) ||
        (292.5 <= angle || angle < 337.5)) {
        if (cacheMagn[ind_y-1][ind_x+1] > centerCell || 
                cacheMagn[ind_y+1][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    }

}	

}
