
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {

__global__
void sobelAndSuppression(int* src, int* dst){

	__shared__ int cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 
 
}


__global__
void transpose_char(int* src, int* dst){

	__shared__ int cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 
 
}
__global__
void transpose_short(int* src, int* dst){
	//printf("WE R IN)");
	__shared__ int cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 
 
}

__global__
void oneDimSobel(int* src, int *dst, int*N){
	int n = N[0]; // width
	int m = N[1]; // height
	int SIZE = n*m;
	int th_x = blockIdx.x*1024+threadIdx.x;
	if(th_x < n){
		int prev = src[th_x];
		int next=-1;
		for(int i=th_x+n; i<SIZE; i+=n){
			next = src[i];
			int temp = next-prev;
			temp = temp>=0?temp:-temp;
			dst[i] = temp;
			prev = next; 
		}
	}
}
__global__
void sobelPlus(int* A, int* B, int* dst){
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	//TU MOZE BYC WOLNO
	int pos = th_y*m+th_x;
	dst[pos] = (A[pos]+B[pos]>20?255:0);
}
/*__global__
void columning(int* cc, int* bitmap, int* N, int* myBool){
	int n =  *N;
	int SIZE =  n * n;
    int th_x = blockIdx.x * 1024 + threadIdx.x;
	//printf("thx = %d\n", th_x);
	if(th_x < n){
		int i = th_x+n;
		//printf("nums %d %d\n", i, i-n);
		for(i = th_x+n; i<SIZE; i+=n){
			if(bitmap[i]==bitmap[i-n] && cc[i-n]<cc[i]){
				cc[i] = cc[i-n];
				*myBool = 1;
			}
		}
		i-=n;
		for(i = i-n ;i>=0; i-=n){
			if(bitmap[i]==bitmap[i+n] && cc[i+n]<cc[i]){
				cc[i]=cc[i+n];
				*myBool = 1;
			}
		}
	}
}*/
}


