
#include <hip/hip_runtime.h>
#include<cstdio>
#define S 64
#define ZERO 0
#define PI 3.14159265
#define LOW 9
#define HIGH 18
#define QUEUE_SIZE 128
#define KERNEL_RADIUS 8

extern "C" {
__device__
bool btwn(int a, int x, int y){
	return (a>=x && a<y);
}

__device__
void load_to_shared(int* src, int cache[][S], int th_x, int th_y, int n, int m){
	int val, pos, ind_x, ind_y;
	if(threadIdx.x==0 && threadIdx.y==0){
		ind_x = th_x-1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x] = val;
	}
	
	if(threadIdx.x==0 && threadIdx.y==31){
		ind_x = th_x-1; ind_y = th_y+1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==0){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+2] = val;
	}
	
	if(threadIdx.x==31 && threadIdx.y==31){
		ind_x = th_x+1; ind_y = th_y-1;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+2] = val;
	}

	if(threadIdx.y==0){
		ind_x = th_x; ind_y = th_y-1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y][threadIdx.x+1] = val;
	}

	if(threadIdx.y==31){
		ind_x = th_x; ind_y = th_y+1; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
		cache[threadIdx.y+2][threadIdx.x+1] = val;
	}
	
	if(threadIdx.x==0){
		ind_x = th_x-1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x] = val;
	}

	if(threadIdx.x==31){
		ind_x = th_x+1; ind_y = th_y; val=ZERO;
		if(btwn(ind_x, 0, m) && btwn(ind_y, 0, n)){
			pos = ind_y*m + ind_x;
			val = src[pos];
		}
	 	cache[threadIdx.y+1][threadIdx.x+2] = val;
	}
}

__global__
void sobel(int* src, int* dstMagni, float * arcTangensOut){
	__shared__ int cache[34][S];
	int m = gridDim.x*32, n = gridDim.y*32, 
	th_x = blockIdx.x * 32 + threadIdx.x, 
	th_y = blockIdx.y * 32 + threadIdx.y,
	i_src = th_y*m + th_x, ind_x, ind_y,
	magn_x, magn_y, magnAbs_x, magnAbs_y;

	/*now we load to shared with a frame of thickness eq 1*/
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; 
	ind_x = threadIdx.x+1; 
	__syncthreads();
	magn_x = cache[ind_y][ind_x-1] - cache[ind_y][ind_x+1];
	magnAbs_x = ((magn_x>0) ? magn_x : -magn_x);
	magn_y = cache[ind_y+1][ind_x] - cache[ind_y-1][ind_x];
	magnAbs_y = ((magn_y>0) ? magn_y : -magn_y);
	dstMagni[i_src] = magnAbs_x + magnAbs_y;
	arcTangensOut[i_src] = atan2((float) magn_y,(float) magn_x)
										* 180 / PI;
}

__global__
void nonMaximalSupression(int * magn, float * arcTangens, int * dest) {
	__shared__ int cacheMagn[34][S];
	int m = gridDim.x*32, n = gridDim.y*32,
	th_x = blockIdx.x * 32 + threadIdx.x, 
	th_y = blockIdx.y * 32 + threadIdx.y,
	i_src = th_y*m + th_x, ind_x, ind_y;
	float angle;

	cacheMagn[threadIdx.y+1][threadIdx.x+1] = magn[i_src];
	load_to_shared(magn, cacheMagn, th_x, th_y, n, m);

	ind_y = threadIdx.y+1; ind_x = threadIdx.x+1; 
	__syncthreads();
   angle = arcTangens[i_src];
    if (angle < 0) angle = 360 + angle;
    
    //north && south
    int centerCell = cacheMagn[ind_y][ind_x];
    dest[i_src] = centerCell;
    if ((337.5 <= angle || angle < 22.5) || 
            (157.25 <= angle && angle < 202.5)) {
        if (cacheMagn[ind_y][ind_x+1] > centerCell ||
                cacheMagn[ind_y][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // north-east && south-west 
    else if ((22.5 <= angle && angle < 67.5) ||
            (202.5 <= angle && angle < 247.5)) {
        if (cacheMagn[ind_y-1][ind_x+1] > centerCell || 
                cacheMagn[ind_y+1][ind_x-1] > centerCell) 
                    dest[i_src] = 0;
    } // west && east
    else if ((67.5 <= angle && angle < 112.5) ||
                (247.5 <= angle && angle < 292.5)) {
        if (cacheMagn[ind_y+1][ind_x] > centerCell ||
                cacheMagn[ind_y-1][ind_x] > centerCell) 
                    dest[i_src] = 0;
    } // west-north && east-south
    else if ((112.5 <= angle && angle < 157.5) ||
        (292.5 <= angle || angle < 337.5)) {
        if (cacheMagn[ind_y-1][ind_x-1] > centerCell || 
                cacheMagn[ind_y+1][ind_x+1] > centerCell) 
                    dest[i_src] = 0;
    }

}	

__global__
void prepareBfs(int* src){
	int m = gridDim.x*32;
	int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int val = src[i_src];
	if(val < LOW){
		src[i_src] = 0;
	}
	else if(val >= HIGH){
		src[i_src] = -2;
	}
	else{
		src[i_src] = -1;
	}

}
__global__
void oneBfs(int* src, int* dst, int* changed){
	__shared__ int cache[34][S];
	int m = gridDim.x*32, n = gridDim.y*32,
	th_x = blockIdx.x * 32 + threadIdx.x,
	th_y = blockIdx.y * 32 + threadIdx.y,
	i_src = th_y*m + th_x,
	ind_y = threadIdx.y+1, ind_x = threadIdx.x+1,
	queue[QUEUE_SIZE],
	beg=0, end=0, val,
	procInd_x, procInd_y, x_new, y_new;
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	val = cache[ind_y][ind_x];
	load_to_shared(src, cache, th_x, th_y, n, m);
	__syncthreads();
	
	if(val==-1){
		for(int i=-1; i<2; ++i){
			for(int j=-1; j<2; ++j){
				procInd_x = ind_x+i;
				procInd_y = ind_y+j;
				if(cache[procInd_y][procInd_x]==-2){
					queue[end++] = ind_y;
					queue[end++] = ind_x;
					cache[ind_y][ind_x]=-2;
					*changed=1;
					i=2;
					j=2;
				}
			}
		}
	}
	

	while(beg!=end){
		procInd_y = queue[beg++];
		procInd_x = queue[beg++];
		for(int i=-1; i<2; ++i){
			for(int j=-1; j<2; ++j){
				x_new = procInd_x+i;
				y_new = procInd_y+j;
				if(cache[y_new][x_new]==-1 && btwn(y_new, 1, 33) 
											&& btwn(x_new, 1, 33)){
					queue[end++] = y_new;
					queue[end++] = x_new;
					cache[y_new][x_new]=-2;
				}
			}
		}
	}
	__syncthreads();
	dst[i_src] = cache[ind_y][ind_x];
	
}
__global__ 
void final_battle(int* src){
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int val = src[i_src];
	if(val==-2){
		src[i_src] = 255;
	}
	else{
		src[i_src] = 0;
	}
}

__global__
void gaussianFilter(int * src, int * dest) {
    __shared__ int cache[34][S];
    int n = gridDim.y*32;
    int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	int ind_y, ind_x;
	cache[threadIdx.y+1][threadIdx.x+1] = src[i_src];
	load_to_shared(src, cache, th_x, th_y, n, m);

    ind_y = threadIdx.y+1+KERNEL_RADIUS; ind_x = threadIdx.x+1+KERNEL_RADIUS;
	__syncthreads();

    int sum = 0;
    for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++) {
        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) {
            sum += cache[ind_y+i][ind_x+j]; //d_kernel[KERNEL_RADIUS + j]
        }
    }
    dest[i_src] = (int) sum;
}

}
