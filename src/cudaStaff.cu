
#include <hip/hip_runtime.h>
#include<cstdio>

extern "C" {
__global__
void transpose_char(unsigned char* src, unsigned char* dst){

	__shared__ unsigned char cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 
 
}
__global__
void transpose_short(short* src, short* dst){
	//printf("WE R IN)");
	__shared__ short cache[32][33];
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	int i_src = th_y*m + th_x;
	cache[threadIdx.y][threadIdx.x] = src[i_src];
    __syncthreads();
    th_x = blockIdx.y * 32 + threadIdx.x;
	th_y = blockIdx.x * 32 + threadIdx.y;
    m = gridDim.y*32;
    int i_dst = th_y*m+th_x;
    dst[i_dst] = cache[threadIdx.x][threadIdx.y]; 
 
}

__global__
void oneDimSobel(unsigned char *src, short *dst, int*N){
	int n = N[0]; // width
	int m = N[1]; // height
	int SIZE = n*m;
	int th_x = blockIdx.x*1024+threadIdx.x;
	if(th_x < n){
		int prev = src[th_x];
		int next=-1;
		for(int i=th_x+n; i<SIZE; i+=n){
			next = src[i];
			int temp = next-prev;
			temp = temp>=0?temp:-temp;
			dst[i] = temp;
			prev = next; 
		}
	}
}
__global__
void sobelPlus(short* A, short* B, unsigned char* dst){
	int m = gridDim.x*32;
    int th_x = blockIdx.x * 32 + threadIdx.x;
	int th_y = blockIdx.y * 32 + threadIdx.y;
	//TU MOZE BYC WOLNO
	int pos = th_y*m+th_x;
	dst[pos] = A[pos]+B[pos]>32?255:0;
}
/*__global__
void columning(int* cc, int* bitmap, int* N, int* myBool){
	int n =  *N;
	int SIZE =  n * n;
    int th_x = blockIdx.x * 1024 + threadIdx.x;
	//printf("thx = %d\n", th_x);
	if(th_x < n){
		int i = th_x+n;
		//printf("nums %d %d\n", i, i-n);
		for(i = th_x+n; i<SIZE; i+=n){
			if(bitmap[i]==bitmap[i-n] && cc[i-n]<cc[i]){
				cc[i] = cc[i-n];
				*myBool = 1;
			}
		}
		i-=n;
		for(i = i-n ;i>=0; i-=n){
			if(bitmap[i]==bitmap[i+n] && cc[i+n]<cc[i]){
				cc[i]=cc[i+n];
				*myBool = 1;
			}
		}
	}
}*/
}


